#include "hip/hip_runtime.h"
// Copyright (c) 2021, Max von Buelow, GRIS, Technical University of Darmstadt

#include <chrono>
#include <cmath>
#include <cstring>
#include <fstream>
#include <iostream>
#include <limits>
#include <float.h>
#include <stdint.h>
#include <stdio.h>

#include "happly.h"
#include "image.h"
#include "meminf.h"

#include "bvh.h"
#include "types.h"

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#endif

#ifndef __HIPCC__
template <typename T>
T min(T a, T b)
{
	return a < b ? a : b;
}
template <typename T>
T max(T a, T b)
{
	return a > b ? a : b;
}

struct float4 { float x, y, z, w; };

#define __HD__
#define __D__
#else
#define __HD__ __host__ __device__
#define __D__ __device__
#endif

struct RayG {
	float org[3];
	float dir[3];
// 	inline void normalize()
// 	{
// 		float n = sqrt(dir[0] * dir[0] + dir[1] * dir[1] + dir[2] * dir[2]);
// 		for (int i = 0; i < 3; ++i) dir[i] /= n;
// 	}
};
__HD__ inline bool tri_intersect(float &t, float &uu, float &vv, const float *rayorg, const float *raydir, const float *v0, const float *v1, const float *v2)
{
	// from wikipedia moeller trumbore
	const float EPSILON = FLT_EPSILON;

	float e1x = v1[0] - v0[0], e1y = v1[1] - v0[1], e1z = v1[2] - v0[2];
	float e2x = v2[0] - v0[0], e2y = v2[1] - v0[1], e2z = v2[2] - v0[2];

	float hx = raydir[1] * e2z - raydir[2] * e2y;
	float hy = raydir[2] * e2x - raydir[0] * e2z;
	float hz = raydir[0] * e2y - raydir[1] * e2x;

	float a = e1x * hx + e1y * hy + e1z * hz;
	if (a > -EPSILON && a < EPSILON) return false; // This ray is parallel to this triangle.

	float f = 1.0 / a;
	float sx = rayorg[0] - v0[0], sy = rayorg[1] - v0[1], sz = rayorg[2] - v0[2];
	float u = f * (sx * hx + sy * hy + sz * hz);
	if (u < 0.0 || u > 1.0) return false;

	float qx = sy * e1z - sz * e1y;
	float qy = sz * e1x - sx * e1z;
	float qz = sx * e1y - sy * e1x;

	float v = f * (raydir[0] * qx + raydir[1] * qy + raydir[2] * qz);
	if (v < 0.0 || u + v > 1.0) return false;
	uu = u;
	vv = v;

	// At this stage we can compute t to find out where the intersection point is on the line.
	t = f * (e2x * qx + e2y * qy + e2z * qz);
	if (t > EPSILON) return true; // ray intersection
	else return false; // This means that there is a line intersection but not a ray intersection.
}

struct camera {
	float x, y, z;
	float mat[9];
	float fov;
};

struct FaceG {
	uint32_t idx[3];
};
struct Vtx {
	float v[3];
	__HD__ Vtx()
	{}
	__HD__ Vtx(const Vtx &vtx) : v{ vtx.v[0], vtx.v[1], vtx.v[2] }
	{}
	__HD__ Vtx(float a, float b, float c) : v{ a, b, c }
	{}
};
struct VtxExtra {
	float v[3];
	__HD__ VtxExtra()
	{}
	__HD__ VtxExtra(const VtxExtra &vtx) : v{ vtx.v[0], vtx.v[1], vtx.v[2] }
	{}
	__HD__ VtxExtra(float a, float b, float c) : v{ a, b, c }
	{}
};


__D__ void g_mkray(float *rayorg, float *raydir, int x, int y, int w, int h, float ox, float oy, float oz, const float *M, float fov = 3)
{
	float a = w / h; // assuming width > height 
	float Px = (2 * ((x + 0.5) / w) - 1) * tan(fov / 2 * M_PI / 180) * a;
	float Py = (1 - 2 * ((y + 0.5) / h)) * tan(fov / 2 * M_PI / 180);


// 	raydir[0] = Px;
// 	raydir[1] = Py;
// 	raydir[2] = -1;

	float indir[] = { Px, Py, -1 };
	for (int i = 0; i < 3; ++i) {
		float acc = 0;
		for (int j = 0; j < 3; ++j) {
			acc += M[i * 3 + j] * indir[j];
		}
		raydir[i] = acc;
	}

	rayorg[0] = ox;
	rayorg[1] = oy;
	rayorg[2] = oz;
// 	ray.normalize();
}



template <typename T>
__D__ void swap(T &a, T &b)
{
	T t;
	t = a;
	a = b;
	b = t;
}
__D__ void intersect_bounding_planes_native(float &t1, float &t2, float min, float max, int axis, const float *rayorg, const float *raydir)
{
	float dirfrac = 1.f / (axis == 0 ? raydir[0] : axis == 1 ? raydir[1] : raydir[2]);

	float ro = axis == 0 ? rayorg[0] : axis == 1 ? rayorg[1] : rayorg[2];
	t1 = (min - ro) * dirfrac;
	t2 = (max - ro) * dirfrac;
	if (t1 > t2) {
		swap(t1, t2);
	}
}

struct BoundsBVH {
	const float *bounds;
	__D__ BoundsBVH(const float *_bounds) : bounds(_bounds)
	{}
	__D__ void intersect(float &t1l, float &t2l, float &t1r, float &t2r, uint32_t idx, const float *rayorg, const float *raydir) const
	{
		const float *b = bounds + idx * 12;
		float q, w, e, r;
		t1l = FLT_MIN; t2l = FLT_MAX;
		t1r = FLT_MIN; t2r = FLT_MAX;
		for (int axis = 0; axis < 3; ++axis) {
			intersect_bounding_planes_native(q, w, b[axis], b[axis + 3], axis, rayorg, raydir); // TODO early return
			t1l = max(t1l, q);
			t2l = min(t2l, w);
			if (t1l > t2l) break;
		}
		for (int axis = 0; axis < 3; ++axis) {
			intersect_bounding_planes_native(e, r, b[axis + 6], b[axis + 9], axis, rayorg, raydir);
			t1r = max(t1r, e);
			t2r = min(t2r, r);
			if (t1r > t2r) break;
		}
	}
};

struct HitPoint {
	uint32_t idx;
	float u, v;
};
struct LeavesBVH {
	const FaceG *tris;
	const Vtx *vtx;
	int nleafesmax;
	__D__ LeavesBVH(const FaceG *_tris, const Vtx *_vtx, int nleafesmax) : tris(_tris), vtx(_vtx), nleafesmax(nleafesmax)
	{}
	__D__ uint32_t get_off(uint32_t li) const
	{
		return li * nleafesmax;
	}
	__D__ bool intersect(float &t, HitPoint *hitpoint, uint32_t idx, uint32_t nchilds, const float *rayorg, const float *raydir) const
	{
		FaceG f = tris[idx];
		Vtx a = vtx[f.idx[0]];
		Vtx b = vtx[f.idx[1]];
		Vtx c = vtx[f.idx[2]];

		float u, vv;
		if (!tri_intersect(t, u, vv, rayorg, raydir, a.v, b.v, c.v)) return false;

		hitpoint->idx = idx;
		hitpoint->u = u;
		hitpoint->v = vv;

		return true;
	}
};



struct StackEntry3 {
	float t0, t1;
	uint32_t idx, leaves;
	__D__ StackEntry3()
	{}
	__D__ StackEntry3(float _t0, float _t1, uint32_t _idx, uint32_t _leaves) : t0(_t0), t1(_t1), idx(_idx), leaves(_leaves)
	{}
};
template <typename Subtrees, typename Bounds, typename Leaves, int L>
__D__ inline bool trace3(float &t, HitPoint *hitpoint, const Subtrees &subtrees /* subtree sizes */, const Bounds &bounds, const Leaves &leaves, const float *rayorg, const float *raydir)
{
	bool hit = false;
	float t0 = 0, t1 = FLT_MAX;

	StackEntry3 stack[128];

	uint32_t ni = 0, li = 0, top = 0;
	while (1) {
		uint32_t st = subtrees[ni];
		uint32_t axis = st >> 30, left_subtree = st & 0x3fffffffu;
#ifdef __HIPCC__
		__syncthreads();
#endif

		if (axis == 3) {

			uint32_t nn = left_subtree;

			uint32_t off = leaves.get_off(li);
			for (int i = 0; i < nn; ++i) {
				float tt = FLT_MAX;
				HitPoint hitpoint_tmp;
				bool bhit = leaves.intersect(tt, &hitpoint_tmp, off + i, nn, rayorg, raydir);
				bhit &= tt < t;
				if (bhit) {
					t = tt;
					*hitpoint = hitpoint_tmp;
					hit = true;
				}
				t1 = min(t1, t);
			}
		} else {
			uint32_t bi = ni - li;

			uint32_t cl = ni + 1, cr = ni + 1 + left_subtree;
			uint32_t ll = li, lr = li + (left_subtree + 1) / 2;


			// TODO check t0 and t
			float t0l = FLT_MAX, t1l = FLT_MIN, t0r = FLT_MAX, t1r = FLT_MIN;
			bounds.intersect(t0l, t1l, t0r, t1r, bi, rayorg, raydir);
			t0l = max(t0l, t0);
			t1l = min(t1l, t1);
			t0r = max(t0r, t0);
			t1r = min(t1r, t1);

			if (t0l > t1l || t0l > t0r) {
				swap(t0l, t0r);
				swap(t1l, t1r);
				swap(cl, cr);
				swap(ll, lr);
			} else {
			}
			if (!(t0r > t1r)) {
				StackEntry3 e = StackEntry3(t0r, t1r, cr, lr);
				stack[top] = e;
				++top;
			} else {
			}

			if (!(t0l > t1l)) {
				t0 = t0l;
				t1 = t1l;
				ni = cl;
				li = ll;
				continue; // don't ascent
			}
		}

		do {
			if (top == 0) {
				return hit;
			}
			--top;
			StackEntry3 e = stack[top];
			ni = e.idx;
			li = e.leaves;
			t0 = e.t0;
			t1 = e.t1;
		} while (t0 > t);
	}
	return false;
}


__D__ void fragment_shader(const float *vin, const float *light, float *colout, bool hit_shadow)
{
	float x = vin[0], y = vin[1], z = vin[2];
	float nx = vin[3], ny = vin[4], nz = vin[5];

	float lx = light[0] - x, ly = light[1] - y, lz = light[2] - z;
	float ll = sqrt(lx * lx + ly * ly + lz * lz);
	if (ll != 0.f) { lx /= ll; ly /= ll; lz /= ll; }

	float dot = fabs(nx * lx + ny * ly + nz * lz);
	colout[0] = min(max(1.f * dot, 0.f), 1.f) - (hit_shadow ? 0.5 : 0);
}


#ifdef __HIPCC__
__global__
#endif
void TraceKernel(int x, int y, uint8_t *framebuf, const uint32_t *subtrees, const float *bounds, const FaceG *faces, const Vtx *vtx, const VtxExtra *ve, uint32_t w, uint32_t h, camera cam, int nleafesmax)
{
#ifdef __HIPCC__
	x = blockDim.x * (0+blockIdx.x) + threadIdx.x; y = blockDim.y * (0+blockIdx.y) + threadIdx.y;
	if (x >= w || y >= h) return;
#endif

	static const float light[] = { 50, 220, 1140 };

	float rayorg[3], raydir[3];
	g_mkray(rayorg, raydir, x, y, w, h, cam.x, cam.y, cam.z, cam.mat, cam.fov/*, 0, 200, 10000*//*, 0, 0, 4000*/);

	float t = FLT_MAX;
	HitPoint hitpoint;

	BoundsBVH bo(bounds);
	LeavesBVH lv(faces, vtx, nleafesmax);

	bool hit = trace3<const uint32_t*, BoundsBVH, LeavesBVH, 5 * 4>(t, &hitpoint, subtrees, bo, lv, rayorg, raydir);

	float res = 1;
	if (hit) {
		float u = hitpoint.u;
		float v = hitpoint.v;
		uint32_t idx = hitpoint.idx;

		FaceG f = lv.tris[idx];

		// load hit vertices completely
		Vtx v0 = lv.vtx[f.idx[0]];
		Vtx v1 = lv.vtx[f.idx[1]];
		Vtx v2 = lv.vtx[f.idx[2]];
		VtxExtra v0e = ve[f.idx[0]];
		VtxExtra v1e = ve[f.idx[1]];
		VtxExtra v2e = ve[f.idx[2]];

		// lerp
		float vertex[6];
		for (int i = 0; i < 3; ++i) {
			vertex[i] = v0.v[i] * (1.f - u - v) + v1.v[i] * u + v2.v[i] * v;
			vertex[3 + i] = v0e.v[i] * (1.f - u - v) + v1e.v[i] * u + v2e.v[i] * v;
		}
		bool hit_shadow = false;

		fragment_shader(vertex, light, &res, hit_shadow);
	}
	framebuf[y * w + x] = res * 255;
}


void trace_gpu_sah(uint8_t *framebuf, uint32_t *subtrees, float *bounds, FaceG *faces, Vtx *vtx, VtxExtra *vtxextra, uint32_t w, uint32_t h, uint32_t maxlvl, camera cam, int nleafesmax)
{
	std::cout << "Max lvl: " << maxlvl << " " << maxlvl * sizeof(StackEntry3) << std::endl;

	std::cout << "Sizes: " << sizeof(FaceG) << " " << sizeof(Vtx) << " " << sizeof(VtxExtra) << std::endl;

#ifdef __HIPCC__
	dim3 blockd(8, 8);
	dim3 gridd((w + blockd.x - 1) / blockd.x, (h + blockd.y - 1) / blockd.y);
	std::cout << "Max lvl: " << maxlvl << " " << maxlvl * sizeof(StackEntry3) << std::endl;

	TraceKernel<<<gridd, blockd>>>(0, 0, framebuf, subtrees, bounds, faces, vtx, vtxextra, w, h, cam, nleafesmax);
#else
	for (int y = 0; y < h; ++y) {
		for (int x = 0; x < w; ++x) {
			TraceKernel(x, y, framebuf, subtrees, bounds, faces, vtx, vtxextra, w, h, cam, nleafesmax);
		}
	}
#endif
}


void *my_malloc(std::size_t s, int desc)
{
	void *p;
#ifdef __HIPCC__
	hipMalloc(&p, s);
#else
	p = malloc(s);
#endif
	meminf_describe(p, desc);
	return p;
}
void my_upload(void *dst, const void *src, std::size_t s)
{
#ifdef __HIPCC__
	hipMemcpy(dst, src, s, hipMemcpyHostToDevice);
#else
	std::memcpy(dst, src, s);
#endif
}
void my_download(void *dst, const void *src, std::size_t s)
{
#ifdef __HIPCC__
	hipMemcpy(dst, src, s, hipMemcpyDeviceToHost);
#else
	std::memcpy(dst, src, s);
#endif
}


int myatoi(const std::string &s)
{
	int v = 1;
	int l = s.size();
	switch (s.back()) {
	case 'm':
	case 'M':
		v *= 1024;
	case 'k':
		v *= 1024;
		--l;
	}
	return v * std::stoi(s.substr(0, l));
}

Mesh loadMesh(const char* name) {
	Mesh mesh;

	happly::PLYData data(name);

	auto& vertex = data.getElement("vertex");
	std::vector<float> x = vertex.getProperty<float>("x");
	std::vector<float> y = vertex.getProperty<float>("y");
	std::vector<float> z = vertex.getProperty<float>("z");

	mesh.vertices.reserve(x.size());
	for (int i = 0; i < x.size(); i++) {
		mesh.vertices.emplace_back(x[i], y[i], z[i]);
	}

	std::vector<std::vector<size_t>> indicesList = data.getFaceIndices<size_t>();
	for (auto& indices : indicesList) {
		// Perform basic triangulation for faces with more than 3 vertices.
		for (int i = 1; i + 1 < indices.size(); i++) {
			mesh.faces.emplace_back(indices[0], indices[i], indices[i+1]);
		}
	}

	return mesh;
}

void trace(const char* name, int x, camera cam, Heuristic heu) {
	image_b test(x, x, 1);
	Mesh mesh = loadMesh(name);
	mesh.compute_normals();


	std::cout << "Mesh statistics: Faces: " << mesh.faces.size() << " Vertices: " << mesh.vertices.size() << std::endl;
	static const int SIZEOF_EXPPATCH = 512;

	BVHBuilder bvhb;

	int nleafesmax = 32;

	int tag = 0;
	if (1||!bvhb.restore(name, tag)) {
		std::vector<float> aabbs(mesh.faces.size() * 6);
		std::vector<float> cens(mesh.faces.size() * 3);
		for (uint32_t i = 0; i < mesh.faces.size(); ++i) {
			const Face &face = mesh.faces[i];
			const Vertex &v0 = mesh.vertices[face.idx[0]];
			const Vertex &v1 = mesh.vertices[face.idx[1]];
			const Vertex &v2 = mesh.vertices[face.idx[2]];
			
			AABB aabb;
			aabb.feed(v0.pos);
			aabb.feed(v1.pos);
			aabb.feed(v2.pos);

			for (int j = 0; j < 3; ++j) {
				aabbs[i * 6 + j] = aabb.min[j];
				aabbs[i * 6 + j + 3] = aabb.max[j];
				cens[i * 3 + j] = (v0.pos[j] + v1.pos[j] + v2.pos[j]) / 3;
			}
		}

		bvhb.construct(cens.data(), aabbs.data(), mesh.faces.size(), nleafesmax, heu);

		bvhb.backup(name, tag);
	} else {
		std::cout << "Found a BVH backup!" << std::endl;
	}

	std::cout << "Got " << bvhb.num_nodes() << " nodes; bounds: " << bvhb.bounds.size() / 4 << " sum: " << bvhb.bounds.size() / 4 << std::endl;

	uint8_t *framebuf = (uint8_t*)my_malloc(test.width() * test.height(), 0);
	uint32_t *d_subtrees = (uint32_t*)my_malloc(bvhb.subtrees.size() * 4, 1);
// 	std::cout << "x" << std::endl;
	my_upload(d_subtrees, (const char*)bvhb.subtrees.data(), bvhb.subtrees.size() * 4 /* TODO: here was a *3 */);
// 	std::cout << "y" << std::endl;
	std::vector<Face> trispermuted(bvhb.leaf_nodes.size());
	float *d_bounds = (float*)my_malloc(bvhb.bounds.size() * 4 * 6, 2);
	my_upload(d_bounds, (const char*)bvhb.bounds.data(), bvhb.bounds.size() * 4 * 6);

	FaceG *d_tris = (FaceG*)my_malloc(trispermuted.size() * 4 * 3, 3);
	Vtx *d_vtx = (Vtx*)my_malloc(mesh.vertices.size() * sizeof(Vtx), 4);
	VtxExtra *d_vtxextra = (VtxExtra*)my_malloc(mesh.vertices.size() * sizeof(VtxExtra), 5);


	std::vector<Vtx> vtx(mesh.vertices.size());
	std::vector<VtxExtra> vtxextra(mesh.vertices.size());

	std::cout << bvhb.leaf_nodes.size() << " " << mesh.faces.size() << " " << bvhb.leaf_nodes.size() * sizeof(Face) << std::endl;
	for (int i = 0; i < bvhb.leaf_nodes.size(); ++i) {
		uint32_t f = bvhb.leaf_nodes[i];
		if (f == -1u) trispermuted[i] = Face(0, 0, 0);
		else trispermuted[i] = mesh.faces[bvhb.leaf_nodes[i]];
// 		std::cout << trispermuted[i].idx[0] << " " << trispermuted[i].idx[1] << " " << trispermuted[i].idx[2] << std::endl;
	}
	for (int i = 0; i < mesh.vertices.size(); ++i) {
		vtx[i] = Vtx(mesh.vertices[i].pos[0], mesh.vertices[i].pos[1], mesh.vertices[i].pos[2]);
		vtxextra[i] = VtxExtra{ mesh.vertices[i].pos[3], mesh.vertices[i].pos[4], mesh.vertices[i].pos[5] };
	}

	my_upload(d_tris, (const char*)trispermuted.data(), trispermuted.size() * 4 * 3);
	my_upload(d_vtx, (const char*)vtx.data(), vtx.size() * sizeof(Vtx));
	my_upload(d_vtxextra, (const char*)vtxextra.data(), vtxextra.size() * sizeof(VtxExtra));

	std::cout << "Starting renderer" << std::endl;

	trace_gpu_sah(framebuf, d_subtrees, d_bounds, d_tris, d_vtx, d_vtxextra, test.width(), test.height(), bvhb.maxlvl, cam, nleafesmax);

	std::cout << "Download" << std::endl;
	my_download((char*)test.data(), framebuf, test.width() * test.height());

	std::cout << "Original mesh size: " << mesh.faces.size() << std::endl;
	std::cout << "Leaf triangles: " << bvhb.leaf_nodes.size() << std::endl;
	image_io::save(test, "test.png");
}

int main(int argc, const char **argv)
{
	camera cam{ std::atof(argv[3]), std::atof(argv[4]), std::atof(argv[5]), {
		std::atof(argv[6]), std::atof(argv[7]), std::atof(argv[8]),
		std::atof(argv[9]), std::atof(argv[10]), std::atof(argv[11]),
		std::atof(argv[12]), std::atof(argv[13]), std::atof(argv[14])
	}, std::atof(argv[15]) };

	int x = myatoi(argv[2]);
	const char *name = argv[1];
	trace(name, x, cam, argv[16] == std::string("sah") ? SAH : MEDIAN);
}


